#include "hip/hip_runtime.h"
//
// Distributed Linear Algebra with Future (DLAF)
//
// Copyright (c) 2018-2021, ETH Zurich
// All rights reserved.
//
// Please, refer to the LICENSE file in the root directory.
// SPDX-License-Identifier: BSD-3-Clause
//

#include <stdio.h>
#include "dlaf/cusolver/assert_info.h"

#ifdef DLAF_ASSERT_ENABLE

#define DLAF_DEFINE_CUSOLVER_ASSERT_INFO(func)                                                 \
  void assertInfo##func(hipStream_t stream, int* info) {                                      \
    dlaf::cusolver::assert_info<<<1, 1, 0, stream>>>(info, [] __device__() { return #func; }); \
  }

#else

#define DLAF_DEFINE_CUSOLVER_ASSERT_INFO(func) \
  void assertInfo##func(hipStream_t stream, int* info) {}

#endif

namespace dlaf {
namespace cusolver {

template <class F>
__global__ void assert_info(int* info, F func) {
  if (*info != 0) {
    printf("Error %s: info != 0 (%d)\n", func(), *info);
    __trap();
  }
}

DLAF_DEFINE_CUSOLVER_ASSERT_INFO(Potrf)
DLAF_DEFINE_CUSOLVER_ASSERT_INFO(Hegst)

}
}
