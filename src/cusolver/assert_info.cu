#include "hip/hip_runtime.h"
//
// Distributed Linear Algebra with Future (DLAF)
//
// Copyright (c) 2018-2022, ETH Zurich
// All rights reserved.
//
// Please, refer to the LICENSE file in the root directory.
// SPDX-License-Identifier: BSD-3-Clause
//

#include <stdio.h>
#include "dlaf/gpu/lapack/assert_info.h"

#ifdef DLAF_ASSERT_ENABLE

#define DLAF_DEFINE_CUSOLVER_ASSERT_INFO(func)                                                      \
  void assertInfo##func(hipStream_t stream, int* info) {                                           \
    dlaf::gpulapack::internal::assert_info<<<1, 1, 0, stream>>>(info,                               \
                                                                [] __device__() { return #func; }); \
  }

#else

#define DLAF_DEFINE_CUSOLVER_ASSERT_INFO(func) \
  void assertInfo##func(hipStream_t stream, int* info) {}

#endif

namespace dlaf::gpulapack::internal {

template <class F>
__global__ void assert_info(int* info, F func) {
  if (*info != 0) {
    printf("Error %s: info != 0 (%d)\n", func(), *info);
#ifdef DLAF_WITH_CUDA
    __trap();
#else
    abort();
#endif
  }
}

DLAF_DEFINE_CUSOLVER_ASSERT_INFO(Potrf)
DLAF_DEFINE_CUSOLVER_ASSERT_INFO(Hegst)

}
