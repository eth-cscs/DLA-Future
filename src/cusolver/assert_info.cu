#include "hip/hip_runtime.h"
//
// Distributed Linear Algebra with Future (DLAF)
//
// Copyright (c) ETH Zurich
// All rights reserved.
//
// Please, refer to the LICENSE file in the root directory.
// SPDX-License-Identifier: BSD-3-Clause
//

#include <cstdio>

#include <whip.hpp>

#include <dlaf/gpu/lapack/assert_info.h>

#ifdef DLAF_ASSERT_ENABLE

#define DLAF_DEFINE_CUSOLVER_ASSERT_INFO(func)                                                      \
  void assertInfo##func(whip::stream_t stream, int* info) {                                         \
    dlaf::gpulapack::internal::assert_info<<<1, 1, 0, stream>>>(info,                               \
                                                                [] __device__() { return #func; }); \
  }

#else

#define DLAF_DEFINE_CUSOLVER_ASSERT_INFO(func) \
  void assertInfo##func(whip::stream_t, int*) {}

#endif

namespace dlaf::gpulapack::internal {

template <class F>
__global__ void assert_info(int* info, F func) {
  if (*info != 0) {
    printf("Error %s: info != 0 (%d)\n", func(), *info);
#ifdef DLAF_WITH_CUDA
    __trap();
#else
    abort();
#endif
  }
}

DLAF_DEFINE_CUSOLVER_ASSERT_INFO(Potrf)
DLAF_DEFINE_CUSOLVER_ASSERT_INFO(Hegst)

}
