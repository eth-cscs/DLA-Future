#include "hip/hip_runtime.h"
//
// Distributed Linear Algebra with Future (DLAF)
//
// Copyright (c) 2018-2022, ETH Zurich
// All rights reserved.
//
// Please, refer to the LICENSE file in the root directory.
// SPDX-License-Identifier: BSD-3-Clause
//

#include "dlaf/eigensolver/tridiag_solver/misc_gpu_kernels.h"
#include "dlaf/gpu/api.h"
#include "dlaf/util_cuda.h"
#include "dlaf/util_math.h"

#include <hip/hip_complex.h>
#include <thrust/execution_policy.h>
#include <thrust/merge.h>

namespace dlaf::eigensolver::internal {

template <class T>
void mergeIndicesOnDevice(const SizeType* begin_ptr, const SizeType* split_ptr, const SizeType* end_ptr,
                          SizeType* out_ptr, const T* v_ptr) {
  auto cmp = [v_ptr] __device__(const SizeType& i1, const SizeType& i2) {
    return v_ptr[i1] < v_ptr[i2];
  };
  // TODO: with Thrust > 1.16 use `thrust::cuda::par_nosync.on(args...)` instead of `thrust::device`
  thrust::merge(thrust::device, begin_ptr, split_ptr, split_ptr, end_ptr, out_ptr, std::move(cmp));
}

DLAF_CUDA_MERGE_INDICES_ETI(, float);
DLAF_CUDA_MERGE_INDICES_ETI(, double);

constexpr unsigned apply_index_sz = 256;

template <class T>
__global__ void applyIndexOnDevice(SizeType len, const SizeType* index_arr, const T* in_arr,
                                   T* out_arr) {
  const SizeType i = blockIdx.x * apply_index_sz + threadIdx.x;
  if (i >= len)
    return;

  out_arr[i] = in_arr[index_arr[i]];
}

template <class T>
void applyIndexOnDevice(SizeType len, const SizeType* index, const T* in, T* out, hipStream_t stream) {
  dim3 nr_threads(apply_index_sz);
  dim3 nr_blocks(util::ceilDiv(to_sizet(len), to_sizet(apply_index_sz)));
  applyIndexOnDevice<<<nr_blocks, nr_threads, 0, stream>>>(len, index, util::cppToCudaCast(in),
                                                           util::cppToCudaCast(out));
}

DLAF_CUDA_APPLY_INDEX_ETI(, float);
DLAF_CUDA_APPLY_INDEX_ETI(, double);

constexpr unsigned cast_complex_kernel_tile_rows = 64;
constexpr unsigned cast_complex_kernel_tile_cols = 16;

template <class T, class CT>
__global__ void castTileToComplex(const unsigned m, const unsigned n, SizeType ld, const T* in,
                                  CT* out) {
  const unsigned i = blockIdx.x * cast_complex_kernel_tile_rows + threadIdx.x;
  const unsigned j = blockIdx.y * cast_complex_kernel_tile_cols + threadIdx.y;

  if (i >= m || j >= n)
    return;

  SizeType idx = i + j * ld;
  if constexpr (std::is_same<T, float>::value) {
    out[idx] = make_hipComplex(in[idx], 0);
  }
  else {
    out[idx] = make_hipDoubleComplex(in[idx], 0);
  }
}

template <class T>
void castTileToComplex(SizeType m, SizeType n, SizeType ld, const T* in, std::complex<T>* out,
                       hipStream_t stream) {
  const unsigned um = to_uint(m);
  const unsigned un = to_uint(n);
  dim3 nr_threads(cast_complex_kernel_tile_rows, cast_complex_kernel_tile_cols);
  dim3 nr_blocks(util::ceilDiv(um, cast_complex_kernel_tile_rows),
                 util::ceilDiv(un, cast_complex_kernel_tile_cols));
  castTileToComplex<<<nr_blocks, nr_threads, 0, stream>>>(um, un, ld, util::cppToCudaCast(in),
                                                          util::cppToCudaCast(out));
}

DLAF_CUDA_CAST_TO_COMPLEX(, float);
DLAF_CUDA_CAST_TO_COMPLEX(, double);

constexpr unsigned invert_index_kernel_sz = 256;

__global__ void invertIndexOnDevice(SizeType len, const SizeType* in, SizeType* out) {
  const SizeType i = blockIdx.x * invert_index_kernel_sz + threadIdx.x;
  if (i >= len)
    return;

  out[in[i]] = i;
}

void invertIndexOnDevice(SizeType len, const SizeType* in, SizeType* out, hipStream_t stream) {
  dim3 nr_threads(invert_index_kernel_sz);
  dim3 nr_blocks(util::ceilDiv(to_sizet(len), to_sizet(invert_index_kernel_sz)));
  invertIndexOnDevice<<<nr_blocks, nr_threads, 0, stream>>>(len, in, out);
}

constexpr unsigned init_index_tile_kernel_sz = 256;

__global__ void initIndexTile(SizeType offset, SizeType len, SizeType* index_arr) {
  const SizeType i = blockIdx.x * init_index_tile_kernel_sz + threadIdx.x;
  if (i >= len)
    return;

  index_arr[i] = i + offset;
}

void initIndexTile(SizeType offset, SizeType len, SizeType* index_arr, hipStream_t stream) {
  dim3 nr_threads(init_index_tile_kernel_sz);
  dim3 nr_blocks(util::ceilDiv(to_uint(len), init_index_tile_kernel_sz));
  initIndexTile<<<nr_blocks, nr_threads, 0, stream>>>(offset, len, index_arr);
}


constexpr unsigned coltype_kernel_sz = 256;

__global__ void setColTypeTile(ColType ct, SizeType len, ColType* ct_arr) {
  const SizeType i = blockIdx.x * init_index_tile_kernel_sz + threadIdx.x;
  if (i >= len)
    return;

  ct_arr[i] = ct;
}

void setColTypeTile(ColType ct, SizeType len, ColType* ct_arr, hipStream_t stream) { 
  dim3 nr_threads(coltype_kernel_sz);
  dim3 nr_blocks(util::ceilDiv(to_uint(len), coltype_kernel_sz));
  setColTypeTile<<<nr_blocks, nr_threads, 0, stream>>>(ct, len, ct_arr);
}

}
