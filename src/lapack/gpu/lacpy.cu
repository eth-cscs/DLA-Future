#include "hip/hip_runtime.h"
//
// Distributed Linear Algebra with Future (DLAF)
//
// Copyright (c) 2018-2022, ETH Zurich
// All rights reserved.
//
// Please, refer to the LICENSE file in the root directory.
// SPDX-License-Identifier: BSD-3-Clause
//

#include "dlaf/gpu/api.h"
#include "dlaf/gpu/assert.cu.h"
#include "dlaf/gpu/blas/api.h"
#include "dlaf/gpu/error.h"
#include "dlaf/lapack/gpu/lacpy.h"
#include "dlaf/types.h"
#include "dlaf/util_cublas.h"
#include "dlaf/util_math.h"

namespace dlaf::gpulapack {
namespace kernels {

using namespace dlaf::util::cuda_operators;

struct LacpyParams {
  static constexpr unsigned kernel_tile_size_rows = 64;
  static constexpr unsigned kernel_tile_size_cols = 16;
};

template <class T>
__device__ void copyAll(const unsigned m, const unsigned n, const T* a, const unsigned lda, T* b,
                        const unsigned ldb) {
  constexpr unsigned kernel_tile_size_rows = LacpyParams::kernel_tile_size_rows;
  constexpr unsigned kernel_tile_size_cols = LacpyParams::kernel_tile_size_cols;

  const unsigned i = blockIdx.x * kernel_tile_size_rows + threadIdx.x;
  const unsigned j = blockIdx.y * kernel_tile_size_cols;

  if (i >= m)
    return;

  const unsigned k_max = min(j + kernel_tile_size_cols, n);

  for (unsigned k = j; k < k_max; ++k)
    b[i + k * ldb] = a[i + k * lda];
}

template <bool (*comp)(unsigned, unsigned), class T>
__device__ void copyDiag(const unsigned m, const unsigned n, const T* a, const unsigned lda, T* b,
                         const unsigned ldb) {
  constexpr unsigned kernel_tile_size_rows = LacpyParams::kernel_tile_size_rows;
  constexpr unsigned kernel_tile_size_cols = LacpyParams::kernel_tile_size_cols;

  const unsigned i = blockIdx.x * kernel_tile_size_rows + threadIdx.x;
  const unsigned j = blockIdx.y * kernel_tile_size_cols;

  if (i >= m)
    return;

  const unsigned k_max = min(j + kernel_tile_size_cols, n);

  for (unsigned k = j; k < k_max; ++k)
    if (comp(i, k))
      b[i + k * ldb] = a[i + k * lda];
}

template <class T>
__global__ void lacpy(hipblasFillMode_t uplo, const unsigned m, const unsigned n, const T* a,
                      const unsigned lda, T* b, const unsigned ldb) {
  constexpr unsigned kernel_tile_size_rows = LacpyParams::kernel_tile_size_rows;
  constexpr unsigned kernel_tile_size_cols = LacpyParams::kernel_tile_size_cols;

  DLAF_GPU_ASSERT_HEAVY(kernel_tile_size_rows % kernel_tile_size_cols == 0);
  DLAF_GPU_ASSERT_HEAVY(kernel_tile_size_rows == blockDim.x);
  DLAF_GPU_ASSERT_HEAVY(1 == blockDim.y);
  DLAF_GPU_ASSERT_HEAVY(1 == blockDim.z);
  DLAF_GPU_ASSERT_HEAVY(gridDim.x == ceilDiv(m, kernel_tile_size_rows));
  DLAF_GPU_ASSERT_HEAVY(gridDim.y == ceilDiv(n, kernel_tile_size_cols));
  DLAF_GPU_ASSERT_HEAVY(1 == gridDim.z);

  const unsigned i = blockIdx.x;
  const unsigned j = blockIdx.y * kernel_tile_size_cols / kernel_tile_size_rows;

  // Note: if (i == j) the kernel tile contains parts of the diagonal

  switch (uplo) {
    case HIPBLAS_FILL_MODE_LOWER:
      if (i == j)
        copyDiag<dlaf::util::isLower>(m, n, a, lda, b, ldb);
      else if (i > j)
        copyAll(m, n, a, lda, b, ldb);
      break;
    case HIPBLAS_FILL_MODE_UPPER:
      if (i == j)
        copyDiag<dlaf::util::isUpper>(m, n, a, lda, b, ldb);
      else if (i < j)
        copyAll(m, n, a, lda, b, ldb);
      break;
    case HIPBLAS_FILL_MODE_FULL:
      // Note: it is more appropriate to use hipMemcpy2DAsync in this case
      DLAF_GPU_ASSERT_HEAVY(false);
      // copyAll(m, n, a, lda, b, ldb);
      break;
  }
}
}

template <class T>
void lacpy(const blas::Uplo uplo, const SizeType m, const SizeType n, const T* a, const SizeType lda,
           T* b, const SizeType ldb, const hipStream_t stream) {
  if (m == 0 || n == 0)
    return;

  DLAF_ASSERT_HEAVY(m <= lda, m, lda);
  DLAF_ASSERT_HEAVY(m <= ldb, m, ldb);

  constexpr unsigned kernel_tile_size_rows = kernels::LacpyParams::kernel_tile_size_rows;
  constexpr unsigned kernel_tile_size_cols = kernels::LacpyParams::kernel_tile_size_cols;

  if (uplo == blas::Uplo::General) {
    const hipMemcpyKind kind = hipMemcpyDefault;
    DLAF_GPU_CHECK_ERROR(hipMemcpy2DAsync(b, to_sizet(ldb) * sizeof(T), a, to_sizet(lda) * sizeof(T),
                                           to_sizet(m) * sizeof(T), to_sizet(n), kind, stream));
  }
  else {
    const unsigned um = to_uint(m);
    const unsigned un = to_uint(n);

    const dim3 nr_threads(kernel_tile_size_rows, 1);
    const dim3 nr_blocks(util::ceilDiv(um, kernel_tile_size_rows),
                         util::ceilDiv(un, kernel_tile_size_cols));
    kernels::lacpy<<<nr_blocks, nr_threads, 0, stream>>>(util::blasToCublas(uplo), um, un,
                                                         util::cppToCudaCast(a), to_uint(lda),
                                                         util::cppToCudaCast(b), to_uint(ldb));
  }
}

DLAF_CUBLAS_LACPY_ETI(, float);
DLAF_CUBLAS_LACPY_ETI(, double);
DLAF_CUBLAS_LACPY_ETI(, std::complex<float>);
DLAF_CUBLAS_LACPY_ETI(, std::complex<double>);
}
