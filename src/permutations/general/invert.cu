#include "hip/hip_runtime.h"
//
// Distributed Linear Algebra with Future (DLAF)
//
// Copyright (c) 2018-2022, ETH Zurich
// All rights reserved.
//
// Please, refer to the LICENSE file in the root directory.
// SPDX-License-Identifier: BSD-3-Clause
//

#pragma once

#include "dlaf/permutations/general/invert.h"

constexpr unsigned invert_index_kernel_sz = 256;

__global__ void invertIndexOnDevice(SizeType len, const SizeType* in, SizeType* out) {
  const SizeType i = blockIdx.x * invert_index_kernel_sz + threadIdx.x;
  if (i >= len)
    return;

  out[in[i]] = i;
}

void invertIndexOnDevice(SizeType len, const SizeType* in, SizeType* out, hipStream_t stream) {
  dim3 nr_threads(invert_index_kernel_sz);
  dim3 nr_blocks(util::ceilDiv(to_sizet(len), to_sizet(invert_index_kernel_sz)));
  invertIndexOnDevice<<<nr_blocks, nr_threads, 0, stream>>>(len, in, out);
}
