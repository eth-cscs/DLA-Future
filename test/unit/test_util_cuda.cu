#include "hip/hip_runtime.h"
//
// Distributed Linear Algebra with Future (DLAF)
//
// Copyright (c) 2018-2022, ETH Zurich
// All rights reserved.
//
// Please, refer to the LICENSE file in the root directory.
// SPDX-License-Identifier: BSD-3-Clause
//

#include "dlaf/gpu/api.h"
#include "dlaf/gpu/error.h"
#include "dlaf/util_cuda.h"

#include <complex>

#include "gtest/gtest.h"
#include "dlaf_test/util_types.h"

using namespace dlaf;
using namespace dlaf::util;
using namespace dlaf::test;
using namespace testing;

using TestTypes = ::testing::Types<float, double>;

template <class T>
class CudaUtilTestHost : public ::testing::Test {};
TYPED_TEST_SUITE(CudaUtilTestHost, TestTypes);

template <class T>
class CudaUtilTestDevice : public ::testing::Test {};
TYPED_TEST_SUITE(CudaUtilTestDevice, TestTypes);

template <class T>
struct cudaComplex;

template <>
struct cudaComplex<float> {
  using Type = hipComplex;
};

template <>
struct cudaComplex<double> {
  using Type = hipDoubleComplex;
};

template <class T>
using cudaComplex_t = typename cudaComplex<T>::Type;

TYPED_TEST(CudaUtilTestHost, CppToCudaCastReal) {
  TypeParam x = 3.55f;

  auto val = cppToCudaCast(x);

  EXPECT_TRUE((std::is_same_v<TypeParam, decltype(val)>) );
  EXPECT_EQ(x, val);

  auto ptr = cppToCudaCast(&x);

  EXPECT_TRUE((std::is_same_v<TypeParam*, decltype(ptr)>) );
  EXPECT_EQ(&x, ptr);
}

TYPED_TEST(CudaUtilTestHost, CppToCudaCastComplex) {
  using T = TypeParam;

  T x = 3.55f;
  T y = -2.35f;

  std::complex<T> z(x, y);

  auto val = cppToCudaCast(z);

  EXPECT_TRUE((std::is_same_v<cudaComplex_t<T>, decltype(val)>) );
  EXPECT_EQ(z.real(), val.x);
  EXPECT_EQ(z.imag(), val.y);

  auto ptr = cppToCudaCast(&z);

  EXPECT_TRUE((std::is_same_v<cudaComplex_t<T>*, decltype(ptr)>) );
  EXPECT_EQ(reinterpret_cast<void*>(&z), reinterpret_cast<void*>(ptr));
  EXPECT_EQ(reinterpret_cast<T*>(&z), &(ptr->x));
  EXPECT_EQ(reinterpret_cast<T*>(&z) + 1, &(ptr->y));
}

TEST(CudaUtilTestHost, CudaOperatorsUnsigned) {
  using namespace cuda_operators;
  using T = unsigned;

  EXPECT_EQ(0u, ceilDiv(0u, 1u));
  EXPECT_EQ(0u, ceilDiv(0u, 10u));
  EXPECT_EQ(3u, ceilDiv(3u, 1u));
  EXPECT_EQ(1u, ceilDiv(3u, 3u));
  EXPECT_EQ(2u, ceilDiv(4u, 3u));
  EXPECT_EQ(2u, ceilDiv(5u, 3u));
  EXPECT_EQ(2u, ceilDiv(6u, 3u));
}

TYPED_TEST(CudaUtilTestHost, CudaOperatorsReal) {
  using namespace cuda_operators;
  using T = TypeParam;

  const T a = 3.55f;
  const T b = 2.15f;
  const T c = -7.65f;

  EXPECT_EQ(a, conj(a));
  EXPECT_EQ(b, real(b));
  EXPECT_EQ(T{0.f}, imag(c));

  EXPECT_NEAR(a * b + c, cuda_operators::fma(a, b, c), 5 * TypeUtilities<T>::error);
}

#define SET(c, real, imag) \
  do {                     \
    (c).x = (real);        \
    (c).y = (imag);        \
  } while (0)

#define EXPECT_EQ_COMPLEX(real, imag, val) \
  do {                                     \
    EXPECT_EQ(real, (val).x);              \
    EXPECT_EQ(imag, (val).y);              \
  } while (0)

#define EXPECT_NEAR_COMPLEX(real, imag, val, error) \
  do {                                              \
    EXPECT_NEAR(real, (val).x, error);              \
    EXPECT_NEAR(imag, (val).y, error);              \
  } while (0)

TYPED_TEST(CudaUtilTestHost, CudaOperatorsComplex) {
  using namespace cuda_operators;
  using T = TypeParam;
  using ComplexT = cudaComplex_t<T>;

  const ComplexT a = cppToCudaCast(std::complex<T>(3.55f, -2.35f));
  const ComplexT b = cppToCudaCast(std::complex<T>(2.15f, 0.66f));
  const ComplexT c = cppToCudaCast(std::complex<T>(-7.65f, -5.12f));
  const T d = 7.77f;

  // The equality operator test is designed requiring a.x != b.x and a.y != b.y.
  ASSERT_TRUE(a.x != b.x && a.y != b.y);

  ComplexT tmp;

  EXPECT_TRUE(a == a);
  EXPECT_FALSE(a != a);

  SET(tmp, a.x, a.y);
  EXPECT_TRUE(a == tmp);
  EXPECT_FALSE(a != tmp);

  SET(tmp, b.x, a.y);
  EXPECT_FALSE(a == tmp);
  EXPECT_TRUE(a != tmp);

  SET(tmp, a.x, b.y);
  EXPECT_FALSE(a == tmp);
  EXPECT_TRUE(a != tmp);

  EXPECT_FALSE(a == b);
  EXPECT_TRUE(a != b);

  EXPECT_EQ_COMPLEX(-(a.x), -(a.y), -a);
  EXPECT_EQ_COMPLEX(a.x, -(a.y), conj(a));
  EXPECT_EQ(b.x, real(b));
  EXPECT_EQ(c.y, imag(c));

  EXPECT_NEAR_COMPLEX(a.x + b.x, a.y + b.y, a + b, 5 * TypeUtilities<T>::error);
  EXPECT_NEAR_COMPLEX(a.x - b.x, a.y - b.y, a - b, 5 * TypeUtilities<T>::error);

  EXPECT_NEAR_COMPLEX(a.x * c.x - a.y * c.y, a.y * c.x + a.x * c.y, a * c, 5 * TypeUtilities<T>::error);
  const T c2 = c.x * c.x + c.y * c.y;
  EXPECT_NEAR_COMPLEX((a.x * c.x + a.y * c.y) / c2, (a.y * c.x - a.x * c.y) / c2, a / c,
                      5 * TypeUtilities<T>::error);

  EXPECT_NEAR_COMPLEX((a * b + c).x, (a * b + c).y, fma(a, b, c), 5 * TypeUtilities<T>::error);

  EXPECT_NEAR_COMPLEX(d * a.x, d * a.y, d * a, 5 * TypeUtilities<T>::error);
  EXPECT_NEAR_COMPLEX(b.x * d, b.y * d, b * d, 5 * TypeUtilities<T>::error);
  EXPECT_NEAR_COMPLEX(c.x / d, c.y / d, c / d, 5 * TypeUtilities<T>::error);
}

__global__ void testOperatorsUnsigned(unsigned* result) {
  using namespace cuda_operators;
  result[0] = ceilDiv(0u, 1u);
  result[1] = ceilDiv(0u, 10u);
  result[2] = ceilDiv(3u, 1u);
  result[3] = ceilDiv(3u, 3u);
  result[4] = ceilDiv(4u, 3u);
  result[5] = ceilDiv(5u, 3u);
  result[6] = ceilDiv(6u, 3u);
}

TEST(CudaUtilTestDevice, CudaOperatorsUnsigned) {
  using namespace cuda_operators;
  using T = unsigned;

  constexpr unsigned res_size = 7;

  T* res_d;
  DLAF_GPU_CHECK_ERROR(hipMalloc(&res_d, res_size * sizeof(T)));

  testOperatorsUnsigned<<<1, 1>>>(res_d);

  T* res_h;
  DLAF_GPU_CHECK_ERROR(hipHostMalloc(&res_h, res_size * sizeof(T)));
  DLAF_GPU_CHECK_ERROR(hipMemcpy(res_h, res_d, res_size * sizeof(T), hipMemcpyDefault));

  EXPECT_EQ(0u, res_h[0]);
  EXPECT_EQ(0u, res_h[1]);
  EXPECT_EQ(3u, res_h[2]);
  EXPECT_EQ(1u, res_h[3]);
  EXPECT_EQ(2u, res_h[4]);
  EXPECT_EQ(2u, res_h[5]);
  EXPECT_EQ(2u, res_h[6]);
}

template <class T>
__global__ void testOperatorsReal(T a, T b, T c, T* result) {
  using namespace cuda_operators;
  result[0] = conj(a);
  result[1] = real(b);
  result[2] = imag(c);

  result[3] = cuda_operators::fma(a, b, c);
}

TYPED_TEST(CudaUtilTestDevice, CudaOperatorsReal) {
  using T = TypeParam;

  const T a = 3.55f;
  const T b = 2.15f;
  const T c = -7.65f;

  constexpr unsigned res_size = 4;

  T* res_d;
  DLAF_GPU_CHECK_ERROR(hipMalloc(&res_d, res_size * sizeof(T)));

  testOperatorsReal<<<1, 1>>>(a, b, c, res_d);

  T* res_h;
  DLAF_GPU_CHECK_ERROR(hipHostMalloc(&res_h, res_size * sizeof(T)));
  DLAF_GPU_CHECK_ERROR(hipMemcpy(res_h, res_d, res_size * sizeof(T), hipMemcpyDefault));

  EXPECT_EQ(a, res_h[0]);
  EXPECT_EQ(b, res_h[1]);
  EXPECT_EQ(T{0.f}, res_h[2]);

  EXPECT_NEAR(a * b + c, res_h[3], 5 * TypeUtilities<T>::error);

  DLAF_GPU_CHECK_ERROR(hipHostFree(res_h));
  DLAF_GPU_CHECK_ERROR(hipFree(res_d));
}

template <class T>
__global__ void testOperatorsComplex(cudaComplex_t<T> a, cudaComplex_t<T> b, cudaComplex_t<T> c, T d,
                                     bool* result_bool, cudaComplex_t<T>* result, T* result_real) {
  using namespace cuda_operators;
  cudaComplex_t<T> tmp;
  result_bool[0] = a == a;
  result_bool[1] = a != a;

  SET(tmp, a.x, a.y);
  result_bool[2] = a == tmp;
  result_bool[3] = a != tmp;

  SET(tmp, b.x, a.y);
  result_bool[4] = a == tmp;
  result_bool[5] = a != tmp;

  SET(tmp, a.x, b.y);
  result_bool[6] = a == tmp;
  result_bool[7] = a != tmp;

  result_bool[8] = a == b;
  result_bool[9] = a != b;

  result[0] = -a;
  result[1] = conj(a);
  result_real[0] = real(b);
  result_real[1] = imag(c);

  result[2] = a + b;
  result[3] = a - b;
  result[4] = a * c;
  result[5] = a / c;

  result[6] = fma(a, b, c);

  result[7] = d * a;
  result[8] = b * d;
  result[9] = c / d;
}

TYPED_TEST(CudaUtilTestDevice, CudaOperatorsComplex) {
  using namespace cuda_operators;
  using T = TypeParam;
  using ComplexT = cudaComplex_t<T>;

  const ComplexT a = cppToCudaCast(std::complex<T>(3.55f, -2.35f));
  const ComplexT b = cppToCudaCast(std::complex<T>(2.15f, 0.66f));
  const ComplexT c = cppToCudaCast(std::complex<T>(-7.65f, -5.12f));
  const T d = 7.77f;

  // The equality operator test is designed requiring a.x != b.x and a.y != b.y.
  ASSERT_FALSE(a.x == b.x || a.y == b.y);

  constexpr unsigned res_bool_size = 10;
  constexpr unsigned res_size = 10;
  constexpr unsigned res_real_size = 2;

  bool* res_bool_d;
  DLAF_GPU_CHECK_ERROR(hipMalloc(&res_bool_d, res_bool_size * sizeof(bool)));
  ComplexT* res_d;
  DLAF_GPU_CHECK_ERROR(hipMalloc(&res_d, res_size * sizeof(ComplexT)));
  T* res_real_d;
  DLAF_GPU_CHECK_ERROR(hipMalloc(&res_real_d, res_real_size * sizeof(T)));

  testOperatorsComplex<<<1, 1>>>(a, b, c, d, res_bool_d, res_d, res_real_d);

  bool* res_bool_h;
  DLAF_GPU_CHECK_ERROR(hipHostMalloc(&res_bool_h, res_bool_size * sizeof(bool)));
  DLAF_GPU_CHECK_ERROR(
      hipMemcpy(res_bool_h, res_bool_d, res_bool_size * sizeof(bool), hipMemcpyDefault));
  ComplexT* res_h;
  DLAF_GPU_CHECK_ERROR(hipHostMalloc(&res_h, res_size * sizeof(ComplexT)));
  DLAF_GPU_CHECK_ERROR(hipMemcpy(res_h, res_d, res_size * sizeof(ComplexT), hipMemcpyDefault));
  T* res_real_h;
  DLAF_GPU_CHECK_ERROR(hipHostMalloc(&res_real_h, res_real_size * sizeof(T)));
  DLAF_GPU_CHECK_ERROR(hipMemcpy(res_real_h, res_real_d, res_real_size * sizeof(T), hipMemcpyDefault));

  EXPECT_TRUE(res_bool_h[0]);
  EXPECT_FALSE(res_bool_h[1]);
  EXPECT_TRUE(res_bool_h[2]);
  EXPECT_FALSE(res_bool_h[3]);
  EXPECT_FALSE(res_bool_h[4]);
  EXPECT_TRUE(res_bool_h[5]);
  EXPECT_FALSE(res_bool_h[6]);
  EXPECT_TRUE(res_bool_h[7]);
  EXPECT_FALSE(res_bool_h[8]);
  EXPECT_TRUE(res_bool_h[9]);

  EXPECT_EQ_COMPLEX(-(a.x), -(a.y), res_h[0]);
  EXPECT_EQ_COMPLEX(a.x, -(a.y), res_h[1]);
  EXPECT_EQ(b.x, res_real_h[0]);
  EXPECT_EQ(c.y, res_real_h[1]);

  EXPECT_NEAR_COMPLEX(a.x + b.x, a.y + b.y, res_h[2], 5 * TypeUtilities<T>::error);
  EXPECT_NEAR_COMPLEX(a.x - b.x, a.y - b.y, res_h[3], 5 * TypeUtilities<T>::error);

  EXPECT_NEAR_COMPLEX(a.x * c.x - a.y * c.y, a.y * c.x + a.x * c.y, res_h[4],
                      5 * TypeUtilities<T>::error);
  const T c2 = c.x * c.x + c.y * c.y;
  EXPECT_NEAR_COMPLEX((a.x * c.x + a.y * c.y) / c2, (a.y * c.x - a.x * c.y) / c2, res_h[5],
                      5 * TypeUtilities<T>::error);

  EXPECT_NEAR_COMPLEX((a * b + c).x, (a * b + c).y, res_h[6], 5 * TypeUtilities<T>::error);

  EXPECT_NEAR_COMPLEX(d * a.x, d * a.y, res_h[7], 5 * TypeUtilities<T>::error);
  EXPECT_NEAR_COMPLEX(b.x * d, b.y * d, res_h[8], 5 * TypeUtilities<T>::error);
  EXPECT_NEAR_COMPLEX(c.x / d, c.y / d, res_h[9], 5 * TypeUtilities<T>::error);

  DLAF_GPU_CHECK_ERROR(hipHostFree(res_real_h));
  DLAF_GPU_CHECK_ERROR(hipHostFree(res_h));
  DLAF_GPU_CHECK_ERROR(hipFree(res_real_d));
  DLAF_GPU_CHECK_ERROR(hipFree(res_d));
}
